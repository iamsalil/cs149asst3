#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "cycleTimer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

__device__ __inline__ void
shadePixelSnow(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    const float kCircleMaxAlpha = .5f;
    const float falloffScale = 4.f;

    float normPixelDist = sqrt(pixelDist) / rad;
    rgb = lookupColor(normPixelDist);

    float maxAlpha = .6f + .4f * (1.f-p.z);
    maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
    alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

__device__ __inline__ void
shadePixelNotSnow(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    // simple: each circle has an assigned color
    int index3 = 3 * circleIndex;
    rgb = *(float3*)&(cuConstRendererParams.color[index3]);
    alpha = .5f;

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // for all pixels in the bonding box
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(index, pixelCenterNorm, p, imgPtr);
            imgPtr++;
        }
    }
}

  //////////////////
 // MY FUNCTIONS //
//////////////////
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

#include "circleBoxTest.cu_inl"
__global__ void
kernelFindTileCircleIntersections(int* tileCircleIntersect, int N, int s, int e) {
    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    float tileL = static_cast<float>(blockIdx.y*16) / static_cast<float>(width);
    float tileR = fminf(1.f, static_cast<float>((blockIdx.y+1)*16) / static_cast<float>(width));
    float tileB = static_cast<float>(blockIdx.z*16) / static_cast<float>(height);
    float tileT = fminf(1.f, static_cast<float>((blockIdx.z+1)*16) / static_cast<float>(height));

    int tileIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int baseOffset = tileIndex * N;

    int localCircleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int circleIndex = localCircleIndex + s;
    int circleIndex3 = 3 * circleIndex;

    if (circleIndex < e) {
        float3 p = *(float3*)(&cuConstRendererParams.position[circleIndex3]);
        float rad = cuConstRendererParams.radius[circleIndex];
        tileCircleIntersect[baseOffset + localCircleIndex] = circleInBoxConservative(p.x, p.y, rad, tileL, tileR, tileT, tileB);
    }
}

__global__ void
kernelPrintArr(int* arr, int idx, int N) {
    printf("  > [");
    for (int i = 0; i < N; i++) {
        printf("%d ", arr[idx*N+i]);
    }
    printf("]\n");
}

__device__ int
scan_warp(int* ptr, const unsigned int idx) {
    const unsigned int lane = idx % 32;

    __syncwarp();
    for (int i = 0; i < 5; i++) {
        int shift = 1<<i;
        if (lane >= shift) {
            int tmp1 = ptr[idx - shift];
            int tmp2 = ptr[idx];
            __syncwarp();
            ptr[idx] = tmp1 + tmp2;
            __syncwarp();
        }
    }
    return (lane > 0) ? ptr[idx-1] : 0;
}

__device__ void
scan_block(int* ptr, const unsigned int idx) {
    const unsigned int lane = idx % 32;
    const unsigned int warp_id = idx >> 5;

    int val = scan_warp(ptr, idx);

    if (lane == 31)
        ptr[warp_id] = ptr[idx];
    __syncthreads();

    if (warp_id == 0)
        scan_warp(ptr, idx);
    __syncthreads();

    if (warp_id > 0)
        val = val + ptr[warp_id-1];
    __syncthreads();

    ptr[idx] = val;
}

__device__ int
scan_warp_test(int* ptr, const unsigned int idx, int tileIndex) {
    const unsigned int lane = idx % 32;

    __syncwarp();
    for (int i = 0; i < 5; i++) {
        if ((tileIndex == 2080) && (lane == 0)) {
            printf("%d: [", i);
            for (int j = 0; j < 32; j++) {
                printf("%d ", ptr[j]);
            }
            printf("] --> ");
        }
        int shift = 1<<i;
        if (lane >= shift) {
            int tmp1 = ptr[idx - shift];
            int tmp2 = ptr[idx];
            __syncwarp();
            ptr[idx] = tmp1 + tmp2;
            __syncwarp();
        }
        if ((tileIndex == 2080) && (lane == 0)) {
            printf("[");
            for (int j = 0; j < 32; j++) {
                printf("%d ", ptr[j]);
            }
            printf("]\n");
        }
    }
    return (lane > 0) ? ptr[idx-1] : 0;
}

__global__ void
kernelMultiExclusiveScan_SingleWarp(int* deviceArr, int length) {
    int tileIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int baseOffset = tileIndex * length;
    if (tileIndex == 2080) {
        printf("    > %d %d\n", tileIndex, baseOffset);
        printf("      > %d\n", threadIdx.x);
    }
    deviceArr[baseOffset + threadIdx.x] = scan_warp_test(deviceArr + baseOffset, threadIdx.x, tileIndex);
}

void multiExclusiveScan_SingleWarp(int* deviceArr, int width, int height, int length) {
    printf("  > single warp exclusive scan\n");
    dim3 blockDim(32, 1, 1);
    dim3 gridDim(1, width, height);
    kernelPrintArr<<<1, 1>>>(deviceArr, 2080, 32);
    kernelMultiExclusiveScan_SingleWarp<<<gridDim, blockDim>>>(deviceArr, length);
    kernelPrintArr<<<1, 1>>>(deviceArr, 2080, 32);
}

__global__ void
kernelMultiExclusiveScan_SingleBlock(int* deviceArr, int length) {
    int tileIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int baseOffset = tileIndex * length;

    scan_block(deviceArr + baseOffset, threadIdx.x);
}

void multiExclusiveScan_SingleBlock(int* deviceArr, int width, int height, int length) {
    printf("  > single block exclusive scan\n");
    dim3 blockDim(256, 1, 1);
    dim3 gridDim(1, width, height);
    kernelMultiExclusiveScan_SingleBlock<<<gridDim, blockDim>>>(deviceArr, length);
}

__global__ void
kernelMultiExclusiveScan_MultiBlock(int* deviceArr, int length) {
    int tileIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int blockInTileOffset = blockIdx.x * blockDim.x;
    int baseOffset = tileIndex * length + blockInTileOffset;
    scan_block(deviceArr + baseOffset, threadIdx.x);
}

__global__ void
kernelMultiCopyMemory(int* deviceArr, int* tempData, int width, int height, int length, int tempTileLength, int numBlocksPerTile) {
    int tileX = blockIdx.x * blockDim.x + threadIdx.x;
    int tileY = blockIdx.y * blockDim.y + threadIdx.y;

    if ((tileX >= width) || (tileY >= height))
        return;

    if (blockIdx.z >= numBlocksPerTile)
        return;

    int tileIndex = tileY * width + tileX;
    int baseOffset = tileIndex * length;
    int blockInTileOffset = blockIdx.z * 256 + 255;

    int tempOffset = tileIndex * tempTileLength;

    tempData[tempOffset + blockIdx.z] = deviceArr[baseOffset + blockInTileOffset];
}

__global__ void
kernelAddTempData(int* deviceArr, int* tempData, int width, int height, int length, int tempTileLength) {
    int tileIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int blockInTileOffset = blockIdx.x * blockDim.x;
    int baseOffset = tileIndex * length + blockInTileOffset;
    int tempOffset = tileIndex * tempTileLength;

    deviceArr[baseOffset + threadIdx.x] += tempData[tempOffset + blockIdx.x];
}

void multiExclusiveScan_MultiBlock(int* deviceArr, int width, int height, int length, int N) {
    printf("  > multi block exclusive scan\n");
    int numBlocksPerTile = (N + 255)/256;
    // Part 1 - Do blocks independently
    printf("    > part 1\n");
    dim3 blockDim(256, 1, 1);
    dim3 gridDim(numBlocksPerTile, width, height);
    kernelMultiExclusiveScan_MultiBlock<<<gridDim, blockDim>>>(deviceArr, length);
    if (numBlocksPerTile <= 32) {
        // Part 2 - Add blocks together
        printf("    > part 2\n");
        int* tempData = NULL;
        hipMalloc(&tempData, sizeof(int) * width * height * 32);
        blockDim = dim3(16, 16, 1);
        gridDim = dim3((width + 15)/16, (height + 15/16), numBlocksPerTile);
        kernelMultiCopyMemory<<<gridDim, blockDim>>>(deviceArr, tempData, width, height, length, 32, numBlocksPerTile);
        multiExclusiveScan_SingleWarp(deviceArr, width, height, 32);
        // Part 3 - Add results back in
        printf("    > part 3\n");
        blockDim = dim3(256, 1, 1);
        gridDim = dim3(numBlocksPerTile, width, height);
        kernelAddTempData<<<gridDim, blockDim>>>(deviceArr, tempData, width, height, length, 32);
        hipFree(tempData);
    } else {
        // Part 2 - Add blocks together
        printf("    > part 2\n");
        int* tempData = NULL;
        hipMalloc(&tempData, sizeof(int) * width * height * 256);
        blockDim = dim3(16, 16, 1);
        gridDim = dim3((width + 15)/16, (height + 15/16), numBlocksPerTile);
        kernelMultiCopyMemory<<<gridDim, blockDim>>>(deviceArr, tempData, width, height, length, 256, numBlocksPerTile);
        multiExclusiveScan_SingleBlock(deviceArr, width, height, 256);
        // Part 3 - Add results back in
        printf("    > part 3\n");
        blockDim = dim3(256, 1, 1);
        gridDim = dim3(numBlocksPerTile, width, height);
        kernelAddTempData<<<gridDim, blockDim>>>(deviceArr, tempData, width, height, length, 256);
        hipFree(tempData);
    }
}

__global__ void
kernelMultiExclusiveScanUpsweep(int N, int twoD, int twoDPlus1, int* arr) {
    int blockIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int baseOffset = blockIndex * N;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= INT_MAX / twoDPlus1) {
        int i = index*twoDPlus1;
        if (i < N)
            arr[baseOffset + i+twoDPlus1-1] += arr[baseOffset + i+twoD-1];
    }
}

__global__ void
kernelMultiExclusiveScanMidpoint(int N, int width, int height, int* arr) {
    int blockX = blockIdx.x * blockDim.x + threadIdx.x;
    int blockY = blockIdx.y * blockDim.y + threadIdx.y;
    int blockIndex = blockY * width + blockX;
    int baseOffset = blockIndex * N;
    if ((blockX < width) && (blockY < height)) {
        arr[baseOffset + N-1] = 0;
    }
}

__global__ void
kernelMultiExclusiveScanDownsweep(int N, int twoD, int twoDPlus1, int* arr) {
    int blockIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int baseOffset = blockIndex * N;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= INT_MAX / twoDPlus1) { 
        int i = index*twoDPlus1;
        if (i < N) {
            int t = arr[baseOffset + i+twoD-1];
        arr[baseOffset + i+twoD-1] = arr[baseOffset + i+twoDPlus1-1];
        arr[baseOffset + i+twoDPlus1-1] += t;
        }
    }
}

void multiExclusiveScan(int* deviceArr, int width, int height, int length) {
    // kernelPrintArr<<<1, 1>>>(deviceArr, 2080, length);

    double startTime;
    double endTime;

    dim3 blockDim(256, 1, 1);
    dim3 gridDim;
    
    // Upsweep
    startTime = CycleTimer::currentSeconds();
    int blocks = (length + 255) / 256;
    for (int twoD = 1; twoD <= length/2; twoD*=2) {
        int twoDPlus1 = 2*twoD;
        blocks /= 2;
        if (blocks == 0)
            blocks = 1;
        gridDim = dim3(blocks, width, height);
        kernelMultiExclusiveScanUpsweep<<<gridDim, blockDim>>>(length, twoD, twoDPlus1, deviceArr);
        hipDeviceSynchronize();
    }
    endTime = CycleTimer::currentSeconds();
    printf("  time: %fms\n", 1000*(endTime - startTime));
    // Mid
    startTime = CycleTimer::currentSeconds();
    blockDim = dim3(16, 16);
    gridDim = dim3((width +15)/16, (height + 15)/16);
    kernelMultiExclusiveScanMidpoint<<<gridDim, blockDim>>>(length, width, height, deviceArr);
    hipDeviceSynchronize();
    endTime = CycleTimer::currentSeconds();
    printf("  time: %fms\n", 1000*(endTime - startTime));
    /// Downsweep
    startTime = CycleTimer::currentSeconds();
    blockDim = dim3(256, 1, 1);
    int effectiveLength = 1;
    for (int twoD = length/2; twoD >= 1; twoD /= 2) {
        int twoDPlus1 = 2*twoD;
        blocks = (effectiveLength + 255) / 256;
        gridDim = dim3(blocks, width, height);
        kernelMultiExclusiveScanDownsweep<<<gridDim, blockDim>>>(length, twoD, twoDPlus1, deviceArr);
        hipDeviceSynchronize();
        effectiveLength *= 2;
    }
    endTime = CycleTimer::currentSeconds();
    printf("  time: %fms\n", 1000*(endTime - startTime));

    // kernelPrintArr<<<1, 1>>>(deviceArr, 2080, length);
}

/*
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
void multiExclusiveScanThrust(int* deviceArr, int width, int height, int length) {
    // kernelPrintArr<<<1, 1>>>(deviceArr, 2080, length);
    thrust::device_ptr<int> d_ptr(deviceArr);
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            thrust::exclusive_scan(d_ptr, d_ptr+length, d_ptr);
            d_ptr += length;
        }
    }
    // kernelPrintArr<<<1, 1>>>(deviceArr, 2080, length);
}
*/

__global__ void
kernelMultiFindStepLocs(int* steppingArr, int*  stepLocs, int* numSteps, int N, int s, int e) {
    int tileIndex = blockIdx.z * gridDim.y + blockIdx.y;
    int baseOffset = tileIndex * N;

    int localCircleIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int circleIndex = localCircleIndex + s;

    if (circleIndex < e) {
        int current = steppingArr[baseOffset + localCircleIndex];
        int next = steppingArr[baseOffset + localCircleIndex+1];
        if (next == current+1) {
            stepLocs[baseOffset + current] = circleIndex;
        }
    } else if (circleIndex == e) {
        numSteps[tileIndex] = steppingArr[baseOffset + localCircleIndex];
    }
}

__global__ void
kernelPixelUpdateSnow(int* tileCircleUpdates, int* tileNumCircles, int N) {
    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    if ((imageX >= width) || (imageY >= height))
        return;

    int pixelIdx = imageY * width + imageX;
    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * pixelIdx]);
    float2 pixelCenter = make_float2(
            (0.5f + static_cast<float>(imageX)) / static_cast<float>(width),
            (0.5f + static_cast<float>(imageY)) / static_cast<float>(height));

    int tileIndex = blockIdx.y * gridDim.x + blockIdx.x;
    int baseOffset = tileIndex * N;

    int circleIndex, circleIndex3;
    float3 circlePosition;
    for (int i = 0; i < tileNumCircles[tileIndex]; i++) {
        circleIndex = tileCircleUpdates[baseOffset + i];
        circleIndex3 = 3 * circleIndex;
        circlePosition = *(float3*)(&cuConstRendererParams.position[circleIndex3]);
        shadePixelSnow(circleIndex, pixelCenter, circlePosition, imgPtr);
    }
}

__global__ void
kernelPixelUpdateNotSnow(int* tileCircleUpdates, int* tileNumCircles, int N) {
    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    if ((imageX >= width) || (imageY >= height))
        return;

    int pixelIdx = imageY * width + imageX;
    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * pixelIdx]);
    float2 pixelCenter = make_float2(
            (0.5f + static_cast<float>(imageX)) / static_cast<float>(width),
            (0.5f + static_cast<float>(imageY)) / static_cast<float>(height));

    int tileIndex = blockIdx.y * gridDim.x + blockIdx.x;
    int baseOffset = tileIndex * N;

    int circleIndex, circleIndex3;
    float3 circlePosition;
    for (int i = 0; i < tileNumCircles[tileIndex]; i++) {
        circleIndex = tileCircleUpdates[baseOffset + i];
        circleIndex3 = 3 * circleIndex;
        circlePosition = *(float3*)(&cuConstRendererParams.position[circleIndex3]);
        shadePixelNotSnow(circleIndex, pixelCenter, circlePosition, imgPtr);
    }
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    // printf("Constructing renderer\n");
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;

    tileCircleIntersect = NULL;
    tileCircleUpdates = NULL;
    tileNumCircles = NULL;
}

CudaRenderer::~CudaRenderer() {
    // printf("Deconstructing renderer\n");
    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);

        hipFree(tileCircleIntersect);
        hipFree(tileCircleUpdates);
        hipFree(tileNumCircles);
    }
}

const Image*
CudaRenderer::getImage() {
    // printf("Get image pointer\n");
    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * myImageWidth * myImageHeight,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    // printf("%d\n", numCircles);
    printf("Load scene\n");
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
    // Figuring out circle space allocation
    if (numCircles <= 32-1) {
        circleSpaceAllocated = 32;
    } else if (numCircles <= 256-1) {
        circleSpaceAllocated = 256;
    } else if (numCircles <= 256*256-1) {
        circleSpaceAllocated = 256*((numCircles + 255)/256);
    } else {
        circleSpaceAllocated = 256*256;
    }
    // printf("%d\n", numCircles);
}

void
CudaRenderer::setup() {
    // printf("Setting up\n");
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
        printf("   Max threads per block:   %d\n", deviceProps.maxThreadsPerBlock);
        printf("   Max grid size:   (%d, %d, %d)\n", deviceProps.maxGridSize[0], deviceProps.maxGridSize[1], deviceProps.maxGridSize[2]);
    }
    printf("---------------------------------------------------------\n");
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * myImageWidth * myImageHeight);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Allocating new buffers
    hipMalloc(&tileCircleIntersect, sizeof(int) * nWidthTiles * nHeightTiles * circleSpaceAllocated);
    hipMalloc(&tileCircleUpdates, sizeof(int) * nWidthTiles * nHeightTiles * circleSpaceAllocated);
    hipMalloc(&tileNumCircles, sizeof(int) * nWidthTiles * nHeightTiles);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = myImageWidth;
    params.imageHeight = myImageHeight;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {
    // printf("Allocating image\n");
    if (image)
        delete image;
    image = new Image(width, height);
    myImageWidth = width;
    myImageHeight = height;
    nWidthTiles = (width + 15)/16;
    nHeightTiles = (height + 15)/16;
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {
    // printf("Clearing image\n");
    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (myImageWidth + blockDim.x - 1) / blockDim.x,
        (myImageHeight + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
    // printf("Advancing animation\n");
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();
}

/*
void
CudaRenderer::render() {
    printf("Rendering image\n");
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    kernelRenderCircles<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}
*/

void
CudaRenderer::render() {
    printf("Rendering image %d, %d, %d\n", nWidthTiles, nHeightTiles, circleSpaceAllocated);
    double startTime;
    double endTime;
    dim3 blockDim;
    dim3 gridDim;
    // s = index of first circle rendering this iteration
    // e = index of first circle not rendering this iteration
    for (int s = 0; s < numCircles; s += 256*256-1) {
        int e = (s + 256*256-1 < numCircles) ? s + 256*256-1 : numCircles;
        int numCirclesRendering = e - s;
        printf("rendering %d circles (%d -> %d)\n", numCirclesRendering, s, e);

        // (1) Tile x circle intersection
        startTime = CycleTimer::currentSeconds();
        blockDim = dim3(256, 1, 1);
        gridDim = dim3((numCirclesRendering + 255)/256, nWidthTiles, nHeightTiles);
        kernelFindTileCircleIntersections<<<gridDim, blockDim>>>(tileCircleIntersect, circleSpaceAllocated, s, e);
        hipDeviceSynchronize();
        endTime = CycleTimer::currentSeconds();
        printf("> intersection time: %fms\n", 1000*(endTime - startTime));

        // (2) Exclusive scan
        startTime = CycleTimer::currentSeconds();
        if (numCirclesRendering <= 32-1) {
            multiExclusiveScan_SingleWarp(tileCircleIntersect, nWidthTiles, nHeightTiles, circleSpaceAllocated);
        } else if (numCirclesRendering <= 256-1) {
            multiExclusiveScan_SingleBlock(tileCircleIntersect, nWidthTiles, nHeightTiles, circleSpaceAllocated);
        } else {
            multiExclusiveScan_MultiBlock(tileCircleIntersect, nWidthTiles, nHeightTiles, circleSpaceAllocated, numCirclesRendering);
        }
        hipDeviceSynchronize();
        endTime = CycleTimer::currentSeconds();
        printf("> exclusive scan time: %fms\n", 1000*(endTime - startTime));

        // (3) Which circles to update
        startTime = CycleTimer::currentSeconds();
        blockDim = dim3(256, 1, 1);
        gridDim = dim3((numCirclesRendering + 255)/256, nWidthTiles, nHeightTiles);
        kernelMultiFindStepLocs<<<gridDim, blockDim>>>(tileCircleIntersect, tileCircleUpdates, tileNumCircles, circleSpaceAllocated, s, e);
        hipDeviceSynchronize();
        endTime = CycleTimer::currentSeconds();
        printf("> find updates time: %fms\n", 1000*(endTime - startTime));

        // (4) Update pixels
        startTime = CycleTimer::currentSeconds();
        blockDim = dim3(16, 16);
        gridDim = dim3(nWidthTiles, nHeightTiles);
        if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
            kernelPixelUpdateSnow<<<gridDim, blockDim>>>(tileCircleUpdates, tileNumCircles, circleSpaceAllocated);
        } else {
            kernelPixelUpdateNotSnow<<<gridDim, blockDim>>>(tileCircleUpdates, tileNumCircles, circleSpaceAllocated);
        }
        hipDeviceSynchronize();
        endTime = CycleTimer::currentSeconds();
        printf("> pixel update time: %fms\n", 1000*(endTime - startTime));
    }
}