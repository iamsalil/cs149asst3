#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
__global__ void
exclusive_scan_upsweep_kernel(int N, int two_d, int two_dplus1, int* input, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= INT_MAX / two_dplus1) {
        int i = index*two_dplus1;
        if (i < N)
            result[i+two_dplus1-1] += result[i+two_d-1];
    }
}

__global__ void
exclusive_scan_midstep(int N, int* result) {
    result[N-1] = 0;
}

__global__ void
exclusive_scan_downsweep_kernel(int N, int two_d, int two_dplus1, int* input, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= INT_MAX / two_dplus1) {
        int i = index*two_dplus1;
        if (i < N) {
            int t = result[i+two_d-1];
	    result[i+two_d-1] = result[i+two_dplus1-1];
	    result[i+two_dplus1-1] += t;
	}
    }
}

void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    const int N2 = nextPow2(N);
    const int threadsPerBlock = 512;
    // Upsweep
    printf("upsweep\n");
    int blocks = (N2 + threadsPerBlock - 1) / threadsPerBlock;
    for (int two_d = 1; two_d <= N2/2; two_d*=2) {
        int two_dplus1 = 2*two_d;
	blocks /= 2;
	if (blocks == 0)
            blocks = 1;
	exclusive_scan_upsweep_kernel<<<blocks, threadsPerBlock>>>(N2, two_d, two_dplus1, input, result);
    }
    // Mid
    exclusive_scan_midstep<<<1, 1>>>(N2, result);
    // Downsweep
    int N_eff = 1;
    for (int two_d = N2/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2*two_d;
	blocks = (N_eff + threadsPerBlock - 1) / threadsPerBlock;
	exclusive_scan_downsweep_kernel<<<blocks, threadsPerBlock>>>(N2, two_d, two_dplus1, input, result);
	N_eff *= 2;
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
__global__ void
flag_repeats_kernel(int* input, int N, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N-1) {
        if (input[index] == input[index+1])
            output[index] = 1;
        else
            output[index] = 0;
    } else if (index == N-1)
        output[index] = 0;
}

__global__ void
copy_data_kernel(int* input, int N, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
        output[index] = input[index];
}

__global__ void
find_step_locs_kernel(int* input, int N, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N-1) {
        if (input[index+1] == input[index]+1) {
            output[input[index]] = index;
        }
    }
}

__global__ void
get_val_at(int* input, int i, int* save_to) {
    *save_to = input[i];
}

void view_device_array(int* device_arr, int N) {
    int* local_arr;
    local_arr = (int*) malloc(N*sizeof(int));
    hipMemcpy(local_arr, device_arr, N*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) {
        printf("%d ", local_arr[i]);
    }
    printf("\n");
    free(local_arr);
}

int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    const int threadsPerBlock = 512;
    const int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;

    flag_repeats_kernel<<<blocks, threadsPerBlock>>>(device_input, length, device_output);
    exclusive_scan(device_output, length, device_output);
    copy_data_kernel<<<blocks, threadsPerBlock>>>(device_output, length, device_input); // output --> input
    find_step_locs_kernel<<<blocks, threadsPerBlock>>>(device_input, length, device_output);

    // Get final piece of data
    int* final_val;
    int* device_final_val;
    final_val = (int*) malloc(sizeof(int));
    hipMalloc((void **)&device_final_val, sizeof(int));
    get_val_at<<<1, 1>>>(device_input, length-1, device_final_val);
    hipMemcpy(final_val, device_final_val, sizeof(int), hipMemcpyDeviceToHost);
    int ret_val = *final_val;
    free(final_val);
    hipFree(device_final_val);

    return ret_val;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
 
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
